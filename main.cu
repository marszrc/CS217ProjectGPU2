/***************************************************************************
 *cr
 *cr            (C) Copyright 2007 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ***************************************************************************/

/*
 * C code for creating the Q data structure for fast convolution-based
 * Hessian multiplication for arbitrary k-space trajectories.
 *
 * Inputs:
 * kx - VECTOR of kx values, same length as ky and kz
 * ky - VECTOR of ky values, same length as kx and kz
 * kz - VECTOR of kz values, same length as kx and ky
 * x  - VECTOR of x values, same length as y and z
 * y  - VECTOR of y values, same length as x and z
 * z  - VECTOR of z values, same length as x and y
 * phi - VECTOR of the Fourier transform of the spatial basis
 *      function, evaluated at [kx, ky, kz].  Same length as kx, ky, and kz.
 *
 * recommended g++ options:
 *  -O3 -lm -ffast-math -funroll-all-loops
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>
#include <malloc.h>

#include "parboil.h"

#include "file.h"
#include "computeQ.cu"

int main (int argc, char *argv[])
{
  int numX, numK;		/* Number of X and K values */
  int original_numK;		/* Number of K values in input file */
  float *kx, *ky, *kz;		/* K trajectory (3D vectors) */
  float *x, *y, *z;		/* X coordinates (3D vectors) */
  float *phiR, *phiI;		/* Phi values (complex) */
  float *phiMag;		/* Magnitude of Phi */
  float *Qr, *Qi;		/* Q signal (complex) */
  struct kValues* kVals;

  float *phiR_d, *phiI_d, *phiMag_d;
  float *Qr_d, *Qi_d;
  float *x_d, *y_d, *z_d;

  struct pb_Parameters *params;
  struct pb_TimerSet timers;

  pb_InitializeTimerSet(&timers);

  /* Read command line */
  params = pb_ReadParameters(&argc, argv);
  if ((params->inpFiles[0] == NULL) || (params->inpFiles[1] != NULL))
  {

    fprintf(stderr, "Expecting one input filename\n");
    exit(-1);
  }

  /* Read in data */
  pb_SwitchToTimer(&timers, pb_TimerID_IO);
  inputData(params->inpFiles[0],
	    &original_numK, &numX,
	    &kx, &ky, &kz,
	    &x, &y, &z,
	    &phiR, &phiI);

  /* Reduce the number of k-space samples if a number is given
   * on the command line */
  if (argc < 2)
    numK = original_numK;
  else
  {
    int inputK;
    char *end;
    inputK = strtol(argv[1], &end, 10);
    if (end == argv[1])
  	{
  	  fprintf(stderr, "Expecting an integer parameter\n");
  	  exit(-1);
  	}

    numK = MIN(inputK, original_numK);
  }

  printf("%d pixels in output; %d samples in trajectory; using %d samples\n",
         numX, original_numK, numK);

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  /* Create CPU data structures */
  createDataStructsCPU(numK, numX, &phiMag, &Qr, &Qi);


  pb_SwitchToTimer(&timers, pb_TimerID_COPY);
  //Compute PhiMag
  /* Allocate memory on GPU */
  hipMalloc((void** )&phiR_d, sizeof(float) * numK);
  hipMalloc((void** )&phiI_d, sizeof(float) * numK);
  hipMalloc((void** )&phiMag_d, sizeof(float) * numK);
  hipDeviceSynchronize();

  /* Copying data on GPU */
  hipMemcpy(phiR_d, phiR, sizeof(float) * numK, hipMemcpyHostToDevice);
  hipMemcpy(phiI_d, phiI, sizeof(float) * numK, hipMemcpyHostToDevice);

  /* Set data on GPU */
  hipMemset(phiMag_d, 0, sizeof(float) * numK);
  hipDeviceSynchronize();

  pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

  /* Compute on GPU */
  ComputePhiMagGPU(numK, phiR_d, phiI_d, phiMag_d);
  hipDeviceSynchronize();

  pb_SwitchToTimer(&timers, pb_TimerID_COPY);

  /* Copy GPU data to memory */
  hipMemcpy(phiMag, phiMag_d, sizeof(float) * numK, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  /* Free memory on GPU */
  hipFree(phiMag_d);
  hipFree(phiI_d);
  hipFree(phiR_d);
  hipDeviceSynchronize();
  //ComputePhiMagCPU(numK, phiR, phiI, phiMag);

  pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);

  kVals = (struct kValues*)calloc(numK, sizeof (struct kValues));
  int k;
  for (k = 0; k < numK; k++) {
    kVals[k].Kx = kx[k];
    kVals[k].Ky = ky[k];
    kVals[k].Kz = kz[k];
    kVals[k].PhiMag = phiMag[k];
  }


  pb_SwitchToTimer(&timers, pb_TimerID_COPY);

  //Compute Q
  /* Allocate memory on GPU */
  hipMalloc((void** )&Qr_d, sizeof(float) * numX);
  hipMalloc((void** )&Qi_d, sizeof(float) * numX);
  hipMalloc((void** )&x_d, sizeof(float) * numX);
  hipMalloc((void** )&y_d, sizeof(float) * numX);
  hipMalloc((void** )&z_d, sizeof(float) * numX);
  hipDeviceSynchronize();

  /* Copy data on GPU */
  hipMemcpy(x_d, x, sizeof(float) * numX, hipMemcpyHostToDevice);
  hipMemcpy(y_d, y, sizeof(float) * numX, hipMemcpyHostToDevice);
  hipMemcpy(z_d, z, sizeof(float) * numX, hipMemcpyHostToDevice);

  /* Set data on GPU */
  hipMemset(Qr_d, 0, sizeof(float) * numX);
  hipMemset(Qi_d, 0, sizeof(float) * numX);
  hipDeviceSynchronize();

  pb_SwitchToTimer(&timers, pb_TimerID_KERNEL);

  /* Compute on GPU */
  ComputeQGPU(numK, numX, kVals, x_d, y_d, z_d, Qr_d, Qi_d);
  hipDeviceSynchronize();

  pb_SwitchToTimer(&timers, pb_TimerID_COPY);

  /* Copy GPU data to memory */
  hipMemcpy(Qr, Qr_d, sizeof(float) * numX, hipMemcpyDeviceToHost);
  hipMemcpy(Qi, Qi_d, sizeof(float) * numX, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  /* Free memory on GPU */
  hipFree(z_d);
  hipFree(y_d);
  hipFree(x_d);
  hipFree(Qi_d);
  hipFree(Qr_d);
  hipDeviceSynchronize();
  hipDeviceReset();

  //ComputeQCPU(numK, numX, kVals, x, y, z, Qr, Qi);

  if (params->outFile)
  {
    /* Write Q to file */
    pb_SwitchToTimer(&timers, pb_TimerID_IO);
    outputData(params->outFile, Qr, Qi, numX);
    pb_SwitchToTimer(&timers, pb_TimerID_COMPUTE);
  }


  free (kx);
  free (ky);
  free (kz);
  free (x);
  free (y);
  free (z);
  free (phiR);
  free (phiI);
  free (phiMag);
  free (kVals);
  free (Qr);
  free (Qi);

  pb_SwitchToTimer(&timers, pb_TimerID_NONE);
  pb_PrintTimerSet(&timers);
  pb_FreeParameters(params);

  return 0;
}
